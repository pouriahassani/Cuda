#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

const int N = 1024;


#define CUDA_CHECK_ERROR(call) \
do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void matmul(float* d_A, float* d_B, float* d_C, const int N){
/*To be implemented*/
}


int main(){
    
    // generate the inout array on the host
    float *h_A = (float*)malloc(N * N * sizeof(float));
    float *h_B = (float*)malloc(N * N * sizeof(float));
    srand(time(NULL));
    for(int i=0 ; i< N ; i++){
        for(int j=0 ; j< N ; j++){
            h_A[i * N + j] = rand() / (float)RAND_MAX;
            h_B[i * N + j] = rand() / (float)RAND_MAX;
        }
        
    }
    

    float* h_C = (float*)malloc(N * N * sizeof(float));

    // declare GPU memory pointers
    float* d_A;
    float* d_B;
    float* d_C;
    

    // allocate GPU memory
    hipMalloc((void**)&d_A , N * N * sizeof(float));
    hipMalloc((void**)&d_B , N * N * sizeof(float));
    hipMalloc((void**)&d_C , N * N * sizeof(float));

    // transfer the array to GPU
    hipMemcpy(d_A, h_A, N * N * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float),hipMemcpyHostToDevice);

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);
    // launch kernel
    /*To be implemented*/
    
    CUDA_CHECK_ERROR(hipGetLastError()); // Check for kernel launch errors
    hipDeviceSynchronize();
        // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize events
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy back results from GPU memory to CPU  memory
    hipMemcpy(h_C , d_C , N * N * sizeof(float),hipMemcpyDeviceToHost);

    // free GPU memory allocation
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Print the time spent by GPU kernel
    printf("Time spent by GPU: %f milliseconds\n", milliseconds);

    
    return 0;
}