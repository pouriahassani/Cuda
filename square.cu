#include <stdio.h>
#include <time.h>
#define ARRAY_SIZE 1024
#include <hip/hip_runtime.h>
__global__ void square(float* d_out, float* d_in){
    int idx = threadIdx.x;
    d_out[idx] = d_in[idx]*d_in[idx];
}

int main(){
    
    // generate the inout array on the host
    float h_in[ARRAY_SIZE];
    srand(time(NULL));
    for(int i=0 ; i< ARRAY_SIZE ; i++){
        h_in[i] = float(i);
    }

    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float* d_in;
    float* d_out;

    // allocate GPU memory
    hipMalloc((void**)&d_in,ARRAY_SIZE * sizeof(float));
    hipMalloc((void**)&d_out,ARRAY_SIZE * sizeof(float));

    // transfer the array to GPU
    hipMemcpy(d_in, h_in, ARRAY_SIZE*sizeof(float),hipMemcpyHostToDevice);

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // launch kernel
    square<<<1,ARRAY_SIZE>>>(d_in,d_out);

        // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize events
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy back results from GPU memory to CPU  memory
    hipMemcpy(d_out, h_out, ARRAY_SIZE*sizeof(float),hipMemcpyDeviceToHost);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    // Print the time spent by GPU kernel
    printf("Time spent by GPU: %f milliseconds\n", milliseconds);
    return 0;
}